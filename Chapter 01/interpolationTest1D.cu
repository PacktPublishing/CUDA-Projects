#include "hip/hip_runtime.h"
#include <stdio.h>

texture<float, 1> tex;

/******************/
/* ERROR CHECKING */
/******************/
#define cudaCHECK(ans) { checkAssert((ans), __FILE__, __LINE__); }
inline void checkAssert(hipError_t errorCode, const char *file, int line, bool abort = true)
{
	if (errorCode != hipSuccess)
	{
		fprintf(stderr, "Check assert: %s %s %d\n", hipGetErrorString(errorCode), file, line);
		if (abort) exit(errorCode);
	}
}

/*********************/
/* TEXTURE FILTERING */
/*********************/
__global__ void textureFilteringKernelNerp(const float * __restrict__ d_samples, const float * __restrict__ d_xCoord, const int numInSamples)
{
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;

	float nn;
	int   ind = (int)round(d_xCoord[tidx]);
	if (d_xCoord[tidx] < 0)
		nn = d_samples[0];
	else if (d_xCoord[tidx] > numInSamples - 1)
		nn = d_samples[numInSamples - 1];
	else
		nn = d_samples[ind];

	printf("argument = %f; texture = %f; nearest neighbor = %f\n", d_xCoord[tidx], tex1D(tex, (d_xCoord[tidx]) + 0.5), nn);
}

__global__ void textureFilteringKernelLerp(const float * __restrict__ d_samples, const float * __restrict__ d_xCoord, const int numInSamples)
{
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;

	float ll;
	if (d_xCoord[tidx] < 0)
		ll = d_samples[0];
	else if (d_xCoord[tidx] > numInSamples - 1)
		ll = d_samples[numInSamples - 1];
	else {
		int ind = floor(d_xCoord[tidx]);
		float alpha = d_xCoord[tidx] - ind;
		ll = (1.f - alpha) * d_samples[ind] + alpha * d_samples[ind + 1];
	}

	printf("argument = %f; texture = %f; linear = %f\n", d_xCoord[tidx], tex1D(tex, (d_xCoord[tidx]) + 0.5), ll);
}

void textureFiltering(float *h_samples, float *d_samples, float *d_xCoord, int numInSamples, int numOutSamples) {

	hipArray* d_cudaArrayData = NULL; hipMallocArray(&d_cudaArrayData, &tex.channelDesc, numInSamples, 1);
	hipMemcpyToArray(d_cudaArrayData, 0, 0, h_samples, sizeof(float) * numInSamples, hipMemcpyHostToDevice);
	hipBindTextureToArray(tex, d_cudaArrayData);

	tex.normalized = false;
	tex.addressMode[0] = hipAddressModeClamp;
	//tex.addressMode[0] = hipAddressModeBorder;
	//tex.addressMode[0] = hipAddressModeWrap;
	//tex.addressMode[0] = hipAddressModeMirror;

	tex.filterMode = hipFilterModePoint;
	textureFilteringKernelNerp << <1, numOutSamples >> >(d_samples, d_xCoord, numInSamples);
	cudaCHECK(hipPeekAtLastError());
	cudaCHECK(hipDeviceSynchronize());	printf("\n\n");

	tex.filterMode = hipFilterModeLinear;
	textureFilteringKernelLerp << <1, numOutSamples >> >(d_samples, d_xCoord, numInSamples);
	cudaCHECK(hipPeekAtLastError());
	cudaCHECK(hipDeviceSynchronize());
}

/********/
/* MAIN */
/********/
int main()
{
	// --- Number of samples
	int numInSamples = 5;

	// --- Number of interpolated samples
	int numOutSamples = 7;

	// --- Input data on host and device
	float *h_samples = (float*)malloc(numInSamples * sizeof(float));
	for (int ind = 0; ind < numInSamples; ind++) {
		h_samples[ind] = (float)ind / (float)numInSamples;
		printf("index = %d; datum = %f\n", ind, h_samples[ind]);
	}
	printf("\n\n");
	float* d_samples;		cudaCHECK(hipMalloc(&d_samples, sizeof(float) * numInSamples));
	cudaCHECK(hipMemcpy(d_samples, h_samples, sizeof(float) * numInSamples, hipMemcpyHostToDevice));

	// --- Output sampling
	float *h_xCoord = (float *)malloc(numOutSamples * sizeof(float));
	h_xCoord[0] = -0.6f; h_xCoord[1] = -0.1f; h_xCoord[2] = 0.6f; h_xCoord[3] = 1.5f; h_xCoord[4] = 2.1f; h_xCoord[5] = 2.9f; h_xCoord[6] = 4.7f;
	float *d_xCoord;		cudaCHECK(hipMalloc(&d_xCoord, sizeof(float) * numOutSamples));
	cudaCHECK(hipMemcpy(d_xCoord, h_xCoord, sizeof(float) * numOutSamples, hipMemcpyHostToDevice));

	textureFiltering(h_samples, d_samples, d_xCoord, numInSamples, numOutSamples);

	return 0;
}

